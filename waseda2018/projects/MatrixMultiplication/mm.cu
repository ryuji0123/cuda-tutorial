#include <iostream>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void matrixMultiplicationKernel(int* A,int* B,int* C,int N)
{
	int ROW = blockIdx.y*blockDim.y+threadIdx.y;
	int COL = blockIdx.x*blockDim.x+threadIdx.x;
	float tmp_sum = 0.0f;

	if(ROW < N && COL < N){
		for(int i=0;i<N;i++){
			tmp_sum += A[ROW*N+i] *B[i*N+COL];
		}
	}
	C[ROW*N+COL] = tmp_sum;
}

void matrixMultiplication(int* A,int* B,int* C,int N);

int main()
{
	int N = 16;

	//Host i/o vectors
	int *h_A;
	int *h_B;
	int *h_C;

	//Device i/o vector
	int *d_A;
	int *d_B;
	int *d_C;

	size_t bytes = N*N*sizeof(int);

	h_A = (int*)malloc(bytes);
	h_B = (int*)malloc(bytes);
	h_C = (int*)malloc(bytes);

	hipMalloc(&d_A,bytes);
	hipMalloc(&d_B,bytes);
	hipMalloc(&d_C,bytes);
	
	// Initialize matricies on the host
	for(int i=0;i<N;i++){
		for(int j=0;j<N;j++){
			h_A[i*N+j] = 2;
			h_B[i*N+j] = 3;
		}
	}

	//Copy host vectors to device
	hipMemcpy(d_A,h_A,bytes,hipMemcpyHostToDevice);	
	hipMemcpy(d_B,h_B,bytes,hipMemcpyHostToDevice);

	matrixMultiplication(d_A,d_B,d_C,N);

	hipMemcpy(h_C,d_C,bytes,hipMemcpyDeviceToHost);
	
	//check result
	for(int i=0;i<4;i++){
		for(int j=0;j<4;j++){
			printf(" %d",h_C[i*N+j]);
		}
		printf("\n");
	}
	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	
	//free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	return 0;
}


void matrixMultiplication(int* A,int* B,int* C,int N)
{
	dim3 threadsPerBlock(N,N);
	dim3 blocksPerGrid(1,1);
	if(N*N>512){
		threadsPerBlock.x = 512;
		threadsPerBlock.y = 512;
		blocksPerGrid.x = ceil(int(N)/double(threadsPerBlock.x));
		blocksPerGrid.y = ceil(int(N)/double(threadsPerBlock.y));
	}
	matrixMultiplicationKernel<<<blocksPerGrid,threadsPerBlock>>>(A,B,C,N);
}


